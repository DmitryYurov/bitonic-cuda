#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <random>

#include "../includes/bitonic.cuh"

bool checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "Error encountered: " << hipGetErrorString(err) << std::endl;
    return false;
  }

  return true;
}

template <typename Deleter> using cuda_ptr = std::unique_ptr<int[], Deleter>; /// A smart point to handle cuda-side arrays

/**
 * @brief Allocates an array of integers on the device side and return cuda_ptr to it.
 *
 * @param size The size of the array of integers to allocate
 * @return a smart pointer to the allocated array, empty in case of allocation failure
 */
auto allocate_device_memory(size_t size) {
  auto deleter = [](int *ptr) {
    if (ptr != nullptr)
      hipFree(ptr);
  };

  auto result = cuda_ptr<decltype(deleter)>(nullptr, std::move(deleter));
  int *device_data = nullptr;
  if (checkCudaError(hipMalloc(&device_data, size * sizeof(int)))) {
    result.reset(device_data);
  }

  return result;
}

// runs bitonic sort, measures its performance and compares it to std::sort
void run_sort(std::vector<int> to_sort, const hipDeviceProp_t &device_prop) {
  const size_t data_size = to_sort.size();

  auto device_data = allocate_device_memory(data_size);
  if (device_data == nullptr) {
    std::cerr << "Failed to allocate device memory" << std::endl;
    return;
  }

  if (!checkCudaError(hipMemcpy(device_data.get(), to_sort.data(), data_size * sizeof(int), hipMemcpyHostToDevice))) {
    return;
  }

  const auto cuda_start = std::chrono::high_resolution_clock::now();
  bitonic::sort(device_data.get(), data_size, device_prop);

  hipDeviceSynchronize(); // wait for all CUDA operations to finish
  const auto cuda_end = std::chrono::high_resolution_clock::now();

  auto sort_result = std::vector<int>(data_size, 0);
  hipMemcpy(sort_result.data(), device_data.get(), data_size * sizeof(int), hipMemcpyDeviceToHost);

  // running std::sort for comparison
  const auto cpu_start = std::chrono::high_resolution_clock::now();
  std::sort(to_sort.begin(), to_sort.end());
  const auto cpu_end = std::chrono::high_resolution_clock::now();

  // the results of running both algorithms must be the same
  if (sort_result != to_sort) {
    std::cerr << "Results are different" << std::endl;
    return;
  }

  const auto cuda_time = std::chrono::duration_cast<std::chrono::milliseconds>(cuda_end - cuda_start).count();
  const auto cpu_time = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end - cpu_start).count();
  std::cout << std::format("{:8} kB {:8} ms {:8} ms", data_size / 256U, cuda_time, cpu_time) << std::endl;
}

int main() {
  int device_count = -1;
  if (!checkCudaError(hipGetDeviceCount(&device_count))) {
    return 1;
  }

  if (device_count < 1) {
    std::cerr << "No CUDA devices found" << std::endl;
    return 0;
  }

  hipSetDevice(0); // setting the first available device

  hipDeviceProp_t device_prop{};
  if (!checkCudaError(hipGetDeviceProperties(&device_prop, 0))) {
    return 1;
  }

  std::cout << device_prop.name << std::endl;
  std::cout << "Device capability: " << device_prop.major << "." << device_prop.minor << std::endl;

  static constexpr int min_val = std::numeric_limits<int>::min();
  static constexpr int max_val = std::numeric_limits<int>::max();
  std::random_device rd;
  std::mt19937 gen{rd()};
  auto distr = std::uniform_int_distribution<int>(min_val, max_val);

  // printing header for the performance table
  std::cout << std::format("{:10} | {:10} | {:10}", "Data size", "GPU time", "CPU time") << std::endl;

  for (size_t i = 0U, data_size = 256U; i < 19U; ++i, data_size <<= 1U) {
    std::vector<int> sample_data{};
    sample_data.reserve(data_size);
    for (size_t j = 0; j < data_size; ++j) {
      sample_data.push_back(distr(gen));
    }

    run_sort(std::move(sample_data), device_prop);
  }

  return 0;
}
