#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <random>

#include "bitonic.cuh"

bool checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "Error encountered: " << hipGetErrorString(err) << std::endl;
    return false;
  }

  return true;
}

template <typename Deleter> using cuda_ptr = std::unique_ptr<int[], Deleter>; /// A smart pointer to handle cuda-side arrays

/**
 * @brief Allocates an array of integers on the device side and returns cuda_ptr to it.
 *
 * @param size The size of the array of integers to allocate
 * @return a smart pointer to the allocated array, empty in case of allocation failure
 */
auto allocate_device_memory(size_t size) {
  auto deleter = [](int *ptr) {
    if (ptr != nullptr)
      hipFree(ptr);
  };

  auto result = cuda_ptr<decltype(deleter)>(nullptr, std::move(deleter));
  int *device_data = nullptr;
  if (checkCudaError(hipMalloc(&device_data, size * sizeof(int)))) {
    result.reset(device_data);
  }

  return result;
}

/**
 * @brief Calculates the execution time between two time points in milliseconds
 *
 * @param start The starting time point of the measurement interval
 * @param end The ending time point of the measurement interval
 * @return The duration between start and end in milliseconds as a double value
 */
double exec_time(std::chrono::high_resolution_clock::time_point start, std::chrono::high_resolution_clock::time_point end) {
  using namespace std::chrono;
  const auto time_us = duration_cast<microseconds>(end - start).count();
  return static_cast<double>(time_us) / 1000.0;
}

/**
 * @brief Runs bitonic sort on GPU, measures its performance and compares it to std::sort
 *
 * This function performs the following steps:
 * 1. Allocates device memory and copies input data to GPU
 * 2. Runs bitonic sort on GPU and measures execution time
 * 3. Copies results back to host memory
 * 4. Runs std::sort on CPU for comparison and measures execution time
 * 5. Validates that both sorting results match
 * 6. Prints performance metrics
 *
 * @param to_sort Vector containing integers to sort
 * @param device_prop CUDA device properties used for sorting
 */
void run_sort(std::vector<int> to_sort, const hipDeviceProp_t &device_prop) {
  const size_t data_size = to_sort.size();

  auto device_data = allocate_device_memory(data_size);
  if (device_data == nullptr) {
    std::cerr << "Failed to allocate device memory" << std::endl;
    return;
  }

  if (!checkCudaError(hipMemcpy(device_data.get(), to_sort.data(), data_size * sizeof(int), hipMemcpyHostToDevice))) {
    return;
  }

  const auto cuda_start = std::chrono::high_resolution_clock::now();
  bitonic::sort(device_data.get(), data_size, device_prop);

  hipDeviceSynchronize(); // wait for all CUDA operations to finish
  const auto cuda_end = std::chrono::high_resolution_clock::now();

  auto sort_result = std::vector<int>(data_size, 0);
  hipMemcpy(sort_result.data(), device_data.get(), data_size * sizeof(int), hipMemcpyDeviceToHost);

  // running std::sort for comparison
  const auto cpu_start = std::chrono::high_resolution_clock::now();
  std::sort(to_sort.begin(), to_sort.end());
  const auto cpu_end = std::chrono::high_resolution_clock::now();

  // the results of running both algorithms must be the same
  if (sort_result != to_sort) {
    std::cerr << "Results are different" << std::endl;
    return;
  }

  const auto cuda_time = exec_time(cuda_start, cuda_end);
  const auto cpu_time = exec_time(cpu_start, cpu_end);
  std::cout << std::format("{:8} kB {:8.2f} ms {:8.2f} ms", data_size / 256U, cuda_time, cpu_time) << std::endl;
}

int main() {
  int device_count = -1;
  if (!checkCudaError(hipGetDeviceCount(&device_count))) {
    return 1;
  }

  if (device_count < 1) {
    std::cerr << "No CUDA devices found" << std::endl;
    return 0;
  }

  hipSetDevice(0); // setting the first available device

  hipDeviceProp_t device_prop{};
  if (!checkCudaError(hipGetDeviceProperties(&device_prop, 0))) {
    return 1;
  }

  std::cout << device_prop.name << std::endl;
  std::cout << "Device capability: " << device_prop.major << "." << device_prop.minor << std::endl;

  static constexpr int min_val = std::numeric_limits<int>::min();
  static constexpr int max_val = std::numeric_limits<int>::max();
  std::random_device rd;
  std::mt19937 gen{rd()};
  auto distr = std::uniform_int_distribution<int>(min_val, max_val);

  // printing header for the performance table
  std::cout << std::format("{:10} | {:10} | {:10}", "Data size", "GPU time", "CPU time") << std::endl;

  for (size_t i = 0U, data_size = 256U; i < 19U; ++i, data_size <<= 1U) {
    std::vector<int> sample_data{};
    sample_data.reserve(data_size);
    for (size_t j = 0; j < data_size; ++j) {
      sample_data.push_back(distr(gen));
    }

    run_sort(std::move(sample_data), device_prop);
  }

  return 0;
}
