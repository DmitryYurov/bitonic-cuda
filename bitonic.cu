#include "hip/hip_runtime.h"
#include "bitonic.cuh"

#include <chrono>

namespace {
/**
 * @brief Device-side swap implementation
 */
__device__ void swap(int &lhs, int &rhs) noexcept {
  const int tmp = lhs;
  lhs = rhs;
  rhs = tmp;
}

/**
 * @brief Determines if two values need to be swapped during the sort based on tonic size, index of the left-hand-side element
 *        and the values in the array
 *
 * @param data_index Index in the data array to compare with the tonic size. This is the index of the lhs entry in the
 *                   original data sequence.
 * @param tonic_size the size of the interval being sorted
 * @param lhs Left-hand side value to compare. This value corresponds to a smaller index in the data array, the index itself
 *            is passed with data_index input argument.
 * @param rhs Right-hand side value to compare. This value corresponds to a larger index in the data array.
 * @return True if the values need a swap, false otherwise
 */
__device__ bool needs_swap(size_t data_index, size_t tonic_size, int lhs, int rhs) noexcept {
  // first find out if we are on an ascending (left) or descending (right) slope of the bitonic sequence
  const bool ascending = (data_index & tonic_size) == 0;
  return (ascending && (lhs > rhs) || (!ascending && lhs < rhs));
}

/**
 * @brief Part of the bitonic sort performed on shared data of the thread block.
 *
 * @param data Device-allocated array to be sorted
 * @note Assumes the size of the input array is a power of 2 and equal to the block size.
 * @note Each thread block processes blockDim.x consecutive elements from the input array
 * @note Uses shared memory of size blockDim.x * sizeof(int) for local sorting
 */
__global__ void bitonic_shared(int *data) {
  extern __shared__ int loc_data[];
  const unsigned thr_id = threadIdx.x;
  const unsigned data_id = blockIdx.x * blockDim.x + thr_id;

  loc_data[thr_id] = data[data_id];
  __syncthreads();

  // the outer loop defines the size of the interval (tonic) for sorting the values in ascending or descending order
  for (unsigned k = 2U; k <= blockDim.x; k <<= 1U) {
    // in the inner loop we choose a comparison interval and swap elements with their partners if necessary
    for (unsigned j = k >> 1U; j > 0U; j >>= 1U) {
      // partner_id must always be larger than thr_id, otherwise we could encounter a race of swaps.
      const unsigned partner_id = thr_id | j;
      if (needs_swap(data_id, k, loc_data[thr_id], loc_data[partner_id])) {
        swap(loc_data[thr_id], loc_data[partner_id]);
      }
      __syncthreads();
    }
  }

  data[data_id] = loc_data[thr_id];
}

__global__ void iteration(int *data, unsigned tonic_size, unsigned stride) {
  const unsigned data_id = blockIdx.x * blockDim.x + threadIdx.x;

  const unsigned partner_idx = data_id | stride;
  if (needs_swap(data_id, tonic_size, data[data_id], data[partner_idx])) {
    swap(data[data_id], data[partner_idx]);
  }
}

/**
 * @brief Part of the bitonic sort using global memory for larger arrays or final stages of sorting.
 *        This function handles the steps after the initial shared memory sorting to merge sorted sequences further.
 *
 * @param data Device-allocated array to be sorted
 * @param data_size Size of the array to sort (must be power of 2)
 * @param grid_size Number of thread blocks to launch
 * @param block_size Number of threads per block
 * @note It is assumed that the input data is already sorted up to the tonics of block_size
 */
__host__ void bitonic_global(int *data, unsigned data_size, unsigned grid_size, unsigned block_size) {
  unsigned size = block_size << 1U;
  while (size <= data_size) {
    for (unsigned stride = size >> 1U; stride > 0U; stride >>= 1U) {
      iteration<<<grid_size, block_size>>>(data, size, stride);
    }
    size <<= 1U;
  }
}

// rounds value down to the nearest power of 2 such that the result
// is greater or equal to the input value
// returns 1 if the input is less than 2
unsigned ceil_2(unsigned val) {
  if (val <= 1U) {
    return 1U;
  }

  unsigned res = 2U;
  while (res < val) {
    res <<= 1U;
  }

  return res;
}

// rounds value down to the nearest power of 2 such that the result
// is less or equal to the input value. Never returns zero
unsigned floor_2(unsigned val) {
  const unsigned res = ceil_2(val);

  if (res < 2U) {
    return res;
  }
  return res == val ? res : res >> 1U;
}

unsigned get_block_size(unsigned data_size, unsigned max_threads_per_block, unsigned shared_mem_per_block) {
  // we assume that data_size is already a power of two
  const unsigned limit = std::min(floor_2(max_threads_per_block), floor_2(shared_mem_per_block / sizeof(int)));
  return std::min(limit, data_size);
}
}  // anonymous namespace

namespace bitonic {
__host__ void sort(int *d_data, unsigned data_size, const hipDeviceProp_t &properties) {
  // finding the block size
  const unsigned block_size = get_block_size(data_size, properties.maxThreadsPerBlock, properties.sharedMemPerBlock);
  const unsigned grid_size = data_size / block_size;

  bitonic_shared<<<grid_size, block_size, block_size * sizeof(int)>>>(d_data);
  if (grid_size > 1U) {
    bitonic_global(d_data, data_size, grid_size, block_size);
  }
}
}  // namespace bitonic
